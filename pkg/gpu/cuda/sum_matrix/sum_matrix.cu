#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK_ERROR(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

void initialData(float* ip,int size)
{
  for(int i=0;i<size;i++)
  {
    ip[i]=(float)(rand()&0xffff)/1000.0f;
  }
}

void printDataFront(float* ip,int size)
{
  for(int i=0;i<size;i++)
  {
    printf("%f ", ip[i]);
  }
  printf("......\n");
  printf("-------------------------\n");
}

void initDevice(int devNum)
{
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK_ERROR(hipGetDeviceProperties(&deviceProp,dev));
  printf("Using device %d: %s\n",dev,deviceProp.name);
  CHECK_ERROR(hipSetDevice(dev));

}

// 核函数，每一个线程计算矩阵中的一个元素
__global__ void sumMatrix(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;   // col
    int iy=threadIdx.y+blockDim.y*blockIdx.y;   // row
    int idx=ix+iy*nx;
    if (ix<nx && iy<ny)
    {
        MatC[idx] = MatA[idx]+MatB[idx];
    }
}

//主函数
int main(int argc,char** argv)
{
    //设备初始化
    printf("strating...\n");
    initDevice(0);

    //输入二维矩阵，4096*4096，单精度浮点型。
    int nx = 1<<12;
    int ny = 1<<12;
    int nBytes = nx*ny*sizeof(float);
    int numToPrint = 16;

    //Malloc，开辟主机内存
    float* A_host = (float*)malloc(nBytes);
    float* B_host = (float*)malloc(nBytes);
    float* C_from_gpu = (float*)malloc(nBytes);

    //初始化矩阵
    initialData(A_host, nx*ny);
    printf("A matrix data: \n");
    printDataFront(A_host, numToPrint);

    initialData(B_host, nx*ny);
    printf("B matrix data: \n");
    printDataFront(B_host, numToPrint);

    //cudaMalloc，开辟设备内存
    float* A_dev = NULL;
    float* B_dev = NULL;
    float* C_dev = NULL;
    CHECK_ERROR(hipMalloc((void**)&A_dev, nBytes));
    CHECK_ERROR(hipMalloc((void**)&B_dev, nBytes));
    CHECK_ERROR(hipMalloc((void**)&C_dev, nBytes));

    //输入数据从主机内存拷贝到设备内存
    CHECK_ERROR(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));

    //二维线程块，32×32
    dim3 block(32, 32);
    //二维线程网格，128×128
    dim3 grid((nx-1)/block.x+1, (ny-1)/block.y+1);

    //将核函数放在线程网格中执行
    sumMatrix<<<grid,block>>>(A_dev, B_dev, C_dev, nx, ny);
    CHECK_ERROR(hipDeviceSynchronize());

    //拷贝回结果数据
    CHECK_ERROR(hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost));

    //输出数据
    printf("result data: \n");
    printDataFront(C_from_gpu, numToPrint);

    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
    free(A_host);
    free(B_host);
    free(C_from_gpu);
    hipDeviceReset();
    return 0;
}