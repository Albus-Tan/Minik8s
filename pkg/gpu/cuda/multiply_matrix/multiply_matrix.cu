#include <stdio.h>
#include <hip/hip_runtime.h>
// Matrix size: 50 * 25 & 25 * 50
const int M = 15;
const int N = 10;

#define CHECK_ERROR(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

// Matrix multiply: C = A * B
__global__ void matrix_multiply(int **A, int **B, int **C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int value = 0;
    for (int k = 0; k < N; k++) {
        value += A[i][k] * B[k][j];
    }
    C[i][j] = value;
}

void initDevice(int devNum)
{
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK_ERROR(hipGetDeviceProperties(&deviceProp,dev));
  printf("Using device %d: %s\n",dev,deviceProp.name);
  CHECK_ERROR(hipSetDevice(dev));

}

int main() {
    printf("strating...\n");
    initDevice(0);

    int **A = (int **) malloc(sizeof(int *) * M);
    int **B = (int **) malloc(sizeof(int *) * N);
    int **C = (int **) malloc(sizeof(int *) * M);

    int *data_A = (int *) malloc(sizeof(int) * M * N);
    int *data_B = (int *) malloc(sizeof(int) * M * N);
    int *data_C = (int *) malloc(sizeof(int) * M * M);
    for (int i = 0; i < M * N; i++) {
        data_A[i] = i;
        data_B[i] = i;
    }

    printf("Matrix A is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            printf("%d ", data_A[i * N + j]);
        }
        printf("\n");
    }

    printf("Matrix B is:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M ; j++) {
            printf("%d ", data_B[i * M + j]);
        }
        printf("\n");
    }

    int *dev_data_A;
    int *dev_data_B;
    int *dev_data_C;

    // malloc matrix (size = M*N) in GPU device
    CHECK_ERROR(hipMalloc((void **) &dev_data_A, sizeof(int) * M * N));
    CHECK_ERROR(hipMalloc((void **) &dev_data_B, sizeof(int) * M * N));
    CHECK_ERROR(hipMalloc((void **) &dev_data_C, sizeof(int) * M * M));

    // copy data from host to GPU device
    CHECK_ERROR(hipMemcpy((void *) dev_data_A, (void *) data_A, sizeof(int) * M * N, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy((void *) dev_data_B, (void *) data_B, sizeof(int) * M * N, hipMemcpyHostToDevice));
    // init C
    CHECK_ERROR(hipMemset((void *) dev_data_C, 0, sizeof(int) * M * M));

    for (int i = 0; i < M; i++) {
        A[i] = dev_data_A + i * N;
        C[i] = dev_data_C + i * M;
    }

    for (int i = 0; i < N; i++) {
        B[i] = dev_data_B + i * M;
    }

    int **dev_A;
    int **dev_B;
    int **dev_C;

    CHECK_ERROR(hipMalloc((void **) &dev_A, sizeof(int *) * M));
    CHECK_ERROR(hipMalloc((void **) &dev_B, sizeof(int *) * N));
    CHECK_ERROR(hipMalloc((void **) &dev_C, sizeof(int *) * M));

    CHECK_ERROR(hipMemcpy((void *) dev_A, (void *) A, sizeof(int *) * M, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy((void *) dev_B, (void *) B, sizeof(int *) * N, hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy((void *) dev_C, (void *) C, sizeof(int *) * M, hipMemcpyHostToDevice));

    dim3 threadPerBlock(5, 5);
    dim3 numBlocks(M / threadPerBlock.x, M / threadPerBlock.y);

    matrix_multiply <<<numBlocks, threadPerBlock>>> (dev_A, dev_B, dev_C);

    // copy result to host
    CHECK_ERROR(hipMemcpy((void *) data_C, (void *) dev_data_C, sizeof(int) * M * M, hipMemcpyDeviceToHost));

    // print result:
    printf("The matrix multiply result is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M ; j++) {
            printf("%d ", data_C[i * M + j]);
        }
        printf("\n");
    }
}